#include "hip/hip_runtime.h"
#include "shared.h"

const uint BX = 512;
const uint BY = 1;
const uint BZ = 1;

// const uint TILE_SIZE_H = 4;
// const uint TILE_SIZE_W = 4;

constexpr uint THREAD_TILE_SIZE_H = 4;
constexpr uint THREAD_TILE_SIZE_W = 4;
// constexpr uint MAX_PADDING = 1;

__forceinline__ __device__ uint __pos(
    const uint n, const uint c, const uint h, const uint w,
    const uint N, const uint C, const uint H, const uint W)
{
    return n * C * H * W + c * H * W + h * W + w;
}

template <typename T, typename U, uint K>
__global__ void conv2d_kernel_bwd_din(
    const T *__restrict__ dout,
    const T *__restrict__ input,
    const U *__restrict__ weights,
    // const U *__restrict__ bias,
    T *__restrict__ din,
    const uint N,
    const uint C,
    const uint H,
    const uint W,
    const uint NUM_TILE_H,
    const uint NUM_TILE_W)
{
    // linear_idx = n * C * H * W + c * H * W + h * W + w
    const uint linear_idx = blockIdx.x * blockDim.x + threadIdx.x;

    const uint n = linear_idx / (C * NUM_TILE_H * NUM_TILE_W);
    // n_mod = c * H * W + h_out * W + w_out
    const uint n_mod = linear_idx % (C * NUM_TILE_H * NUM_TILE_W);

    const uint c = (n_mod) / (NUM_TILE_H * NUM_TILE_W);
    // c_mod = h * W + w_out
    const uint c_mod = (n_mod) % (NUM_TILE_H * NUM_TILE_W);

    const uint h_tile_idx = c_mod / NUM_TILE_W;
    const uint w_tile_idx = c_mod % NUM_TILE_W;

    if (n >= N || c >= C)
        return;

    T dout_local[THREAD_TILE_SIZE_H + K - 1][THREAD_TILE_SIZE_W + K - 1] = {static_cast<T>(0)};
    T w_local[K][K];

    T din_local[THREAD_TILE_SIZE_H][THREAD_TILE_SIZE_W];

#pragma unroll
    for (uint r = 0; r < THREAD_TILE_SIZE_H + K - 1; r++)
    {
#pragma unroll
        for (uint s = 0; s < THREAD_TILE_SIZE_W + K - 1; s++)
        {
            const int h_in = h_tile_idx * THREAD_TILE_SIZE_H + r - (K / 2);
            const int w_in = w_tile_idx * THREAD_TILE_SIZE_W + s - (K / 2);

            if (0 <= h_in && h_in < H && 0 <= w_in && w_in < W)
                dout_local[r][s] = dout[__pos(n, c, h_in, w_in, N, C, H, W)];
        }
    }

#pragma unroll
    for (uint r = 0; r < K; r++)
#pragma unroll
        for (uint s = 0; s < K; s++)
        {
            const uint w_idx = c * K * K + r * K + s;
            set_value(&w_local[r][s], weights[w_idx]);
        }

#pragma unroll
    for (uint h_idx = 0; h_idx < THREAD_TILE_SIZE_H; h_idx++)
#pragma unroll
        for (uint w_idx = 0; w_idx < THREAD_TILE_SIZE_W; w_idx++)
        {
            const uint h_out = h_tile_idx * THREAD_TILE_SIZE_H + h_idx;
            const uint w_out = w_tile_idx * THREAD_TILE_SIZE_W + w_idx;

            if (h_out < H && w_out < W)
            {
                T accum{static_cast<T>(0)};

#pragma unroll
                for (int r = 0; r < K; r++)
#pragma unroll
                    for (int s = 0; s < K; s++)
                        accum += dout_local[h_idx + r][w_idx + s] * w_local[K - r - 1][K - s - 1];

                din_local[h_idx][w_idx] = accum;
            }
        }

#pragma unroll
    for (uint h_idx = 0; h_idx < THREAD_TILE_SIZE_H; h_idx++)
#pragma unroll
        for (uint w_idx = 0; w_idx < THREAD_TILE_SIZE_W; w_idx++)
        {
            const uint h_out = h_tile_idx * THREAD_TILE_SIZE_H + h_idx;
            const uint w_out = w_tile_idx * THREAD_TILE_SIZE_W + w_idx;

            if (h_out < H && w_out < W)
                din[__pos(n, c, h_out, w_out, N, C, H, W)] = din_local[h_idx][w_idx];
        }
}

template <typename T, typename U, uint K>
__global__ void conv2d_kernel_bwd_dweights(
    const T *__restrict__ dout,
    const T *__restrict__ input,
    const U *__restrict__ weights,
    // const U *__restrict__ bias,
    T *__restrict__ dweights,
    const uint N,
    const uint C,
    const uint H,
    const uint W,
    const uint NUM_TILE_H,
    const uint NUM_TILE_W)
{
    // linear_idx = n * C * H * W + c * H * W + h * W + w
    const uint linear_idx = blockIdx.x * blockDim.x + threadIdx.x;

    const uint n = linear_idx / (C * NUM_TILE_H * NUM_TILE_W);
    // n_mod = c * H * W + h_out * W + w_out
    const uint n_mod = linear_idx % (C * NUM_TILE_H * NUM_TILE_W);

    const uint c = (n_mod) / (NUM_TILE_H * NUM_TILE_W);
    // c_mod = h * W + w_out
    const uint c_mod = (n_mod) % (NUM_TILE_H * NUM_TILE_W);

    const uint h_tile_idx = c_mod / NUM_TILE_W;
    const uint w_tile_idx = c_mod % NUM_TILE_W;

    if (n >= N || c >= C)
        return;

    T dout_local[THREAD_TILE_SIZE_H + K - 1][THREAD_TILE_SIZE_W + K - 1] = {static_cast<T>(0)};
    T in_local[THREAD_TILE_SIZE_H + K - 1][THREAD_TILE_SIZE_W + K - 1] = {static_cast<T>(0)};

    T dw_local[K][K];

#pragma unroll
    for (uint r = 0; r < THREAD_TILE_SIZE_H + K - 1; r++)
    {
#pragma unroll
        for (uint s = 0; s < THREAD_TILE_SIZE_W + K - 1; s++)
        {
            const int h_in = h_tile_idx * THREAD_TILE_SIZE_H + r - (K / 2);
            const int w_in = w_tile_idx * THREAD_TILE_SIZE_W + s - (K / 2);

            if (0 <= h_in && h_in < H && 0 <= w_in && w_in < W)
            {
                dout_local[r][s] = dout[__pos(n, c, h_in, w_in, N, C, H, W)];
                in_local[r][s] = input[__pos(n, c, h_in, w_in, N, C, H, W)];

                // printf("rs: [%u][%u] value: %f \n", r, s, in_local[r][s]);
            }
        }
    }

#pragma unroll
    for (uint k1 = 0; k1 < K; k1++)
#pragma unroll
        for (uint k2 = 0; k2 < K; k2++)
        {
            T accum{static_cast<T>(0)};

            for (uint r = K / 2; r < THREAD_TILE_SIZE_H + K - 1 - K / 2; r++)
                for (uint s = K / 2; s < THREAD_TILE_SIZE_W + K - 1 - K / 2; s++)
                {
                    const int i = r - (K / 2) + k1;
                    const int j = s - (K / 2) + k2;

                    accum += dout_local[r][s] * in_local[i][j];
                }

            dw_local[k1][k2] = accum;
            // printf("[%u][%u] %f \n", k1, k2, accum);
        }

#pragma unroll
    for (uint h_idx = 0; h_idx < THREAD_TILE_SIZE_H; h_idx++)
#pragma unroll
        for (uint w_idx = 0; w_idx < THREAD_TILE_SIZE_W; w_idx++)
        {
            const uint h_out = h_tile_idx * THREAD_TILE_SIZE_H + h_idx;
            const uint w_out = w_tile_idx * THREAD_TILE_SIZE_W + w_idx;

            if (h_out < H && w_out < W)
            {
#pragma unroll
                for (uint k1 = 0; k1 < K; k1++)
#pragma unroll
                    for (uint k2 = 0; k2 < K; k2++)
                    {
                        const uint pos = c * K * K * N * NUM_TILE_H * NUM_TILE_W +
                                         k1 * K * N * NUM_TILE_H * NUM_TILE_W +
                                         k2 * N * NUM_TILE_H * NUM_TILE_W +
                                         n * NUM_TILE_H * NUM_TILE_W +
                                         h_tile_idx * NUM_TILE_W +
                                         w_tile_idx;

                        dweights[pos] = dw_local[k1][k2];
                    }
            }
        }
}

std::vector<torch::Tensor> conv2d_cuda_nchw_bwd(
    torch::Tensor dout,
    torch::Tensor input,
    torch::Tensor weights,
    // torch::Tensor bias,
    uint padding)
{
    const uint N = input.size(0);
    const uint C = input.size(1);
    const uint H = input.size(2);
    const uint W = input.size(3);

    TORCH_CHECK(N == dout.size(0), "");
    TORCH_CHECK(C == dout.size(1), "");
    TORCH_CHECK(H == dout.size(2), "");
    TORCH_CHECK(W == dout.size(3), "");

    const uint c_weight = weights.size(0);
    const uint should_one = weights.size(1);
    const uint R = weights.size(2);
    const uint S = weights.size(3);

    TORCH_CHECK(C == c_weight, "input.shape[1] should be equal to weights.shape[0]");
    TORCH_CHECK(should_one == 1, "weights.size[1] should be one");
    TORCH_CHECK(R == S, "Kernel size should be square");
    TORCH_CHECK(R % 2 == 1 && S % 2 == 1, "Kernel size should be odd number");
    TORCH_CHECK(2 * padding + 1 == R, "Kernel size should be equal to 2 * padding + 1")

    const uint NUM_TILE_H = (H + THREAD_TILE_SIZE_H - 1U) / THREAD_TILE_SIZE_H;
    const uint NUM_TILE_W = (W + THREAD_TILE_SIZE_W - 1U) / THREAD_TILE_SIZE_W;

    torch::Tensor din = torch::empty({N, C, H, W}, input.options());
    torch::Tensor dweights = torch::zeros({C, 1, R, S, N, NUM_TILE_H, NUM_TILE_W}, input.options());

    dim3 threadsPerBlock(BX, BY, BZ);
    dim3 numBlocks(((N * C * NUM_TILE_H * NUM_TILE_W + BX - 1) / BX), 1, 1);

    if (R == 3)
    {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(input.scalar_type(), weights.scalar_type(),
                                         "depthwise conv2d_nchw bwd din",
                                         ([&]
                                          { conv2d_kernel_bwd_din<input_t, weight_t, 3U><<<numBlocks, threadsPerBlock>>>(
                                                static_cast<input_t *>(dout.data_ptr()),
                                                static_cast<input_t *>(input.data_ptr()),
                                                static_cast<weight_t *>(weights.data_ptr()),
                                                // static_cast<weight_t *>(bias.data_ptr()),
                                                static_cast<input_t *>(din.data_ptr()),
                                                N,
                                                C,
                                                H,
                                                W,
                                                NUM_TILE_H,
                                                NUM_TILE_W); }));

        DISPATCH_FLOAT_AND_HALF_AND_BF16(input.scalar_type(), weights.scalar_type(),
                                         "depthwise conv2d_nchw bwd dweights",
                                         ([&]
                                          { conv2d_kernel_bwd_dweights<input_t, weight_t, 3U><<<numBlocks, threadsPerBlock>>>(
                                                static_cast<input_t *>(dout.data_ptr()),
                                                static_cast<input_t *>(input.data_ptr()),
                                                static_cast<weight_t *>(weights.data_ptr()),
                                                // static_cast<weight_t *>(bias.data_ptr()),
                                                static_cast<input_t *>(dweights.data_ptr()),
                                                N,
                                                C,
                                                H,
                                                W,
                                                NUM_TILE_H,
                                                NUM_TILE_W); }));
    }
    else if (R == 5)
    {
        DISPATCH_FLOAT_AND_HALF_AND_BF16(input.scalar_type(), weights.scalar_type(),
                                         "depthwise conv2d_nchw bwd din",
                                         ([&]
                                          { conv2d_kernel_bwd_din<input_t, weight_t, 5U><<<numBlocks, threadsPerBlock>>>(
                                                static_cast<input_t *>(dout.data_ptr()),
                                                static_cast<input_t *>(input.data_ptr()),
                                                static_cast<weight_t *>(weights.data_ptr()),
                                                // static_cast<weight_t *>(bias.data_ptr()),
                                                static_cast<input_t *>(din.data_ptr()),
                                                N,
                                                C,
                                                H,
                                                W,
                                                NUM_TILE_H,
                                                NUM_TILE_W); }));

        DISPATCH_FLOAT_AND_HALF_AND_BF16(input.scalar_type(), weights.scalar_type(),
                                         "depthwise conv2d_nchw bwd dweights",
                                         ([&]
                                          { conv2d_kernel_bwd_dweights<input_t, weight_t, 5U><<<numBlocks, threadsPerBlock>>>(
                                                static_cast<input_t *>(dout.data_ptr()),
                                                static_cast<input_t *>(input.data_ptr()),
                                                static_cast<weight_t *>(weights.data_ptr()),
                                                // static_cast<weight_t *>(bias.data_ptr()),
                                                static_cast<input_t *>(dweights.data_ptr()),
                                                N,
                                                C,
                                                H,
                                                W,
                                                NUM_TILE_H,
                                                NUM_TILE_W); }));
    }

    return {din, dweights.sum(std::vector<int64_t>({-3, -2, -1}))};
}